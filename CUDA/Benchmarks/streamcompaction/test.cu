#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/random.h>

int main() {

  
  hipEvent_t start, stop;
  // Allocate the host input vector A
  int *h_A = (int*)malloc(512000000 * sizeof(int));


  int value = 0;
  int numElements = 8000000;
  float input = 0.1;
  srand(2014);
    for(int i = 0; i < numElements; i++)
        h_A[i] = value;
    int M = (numElements * input)/100;
    int m = M;
    while(m>0){
        int x = (int)(numElements*(((float)rand()/(float)RAND_MAX)));
        if(h_A[x]==value){
            h_A[x] = x+2;
            m--;
        }
    }
    hipEventCreate(&start);
    hipEventCreate(&stop);






  // Transfer data to the device.
  thrust::device_vector<int> d_vec = h_vec;

  // Sort data on the device.
  thrust::sort(d_vec.begin(), d_vec.end());

  // Transfer data back to host.
  thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
}
