#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <time.h>
#include <stdio.h>
#include <vector>

#define T int
#define WARMUP 2
#define REP 10

struct is_even{
  __host__ __device__
  bool operator()(const T &x){
    return (x % 2) == 0;
  }
};


int main(){
  for (float i = 0.0; i <1.05; i += 0.1){
    int input = i;
    int numElements = 16000000;
    size_t size = numElements * sizeof(T);
    
    int value = 0;
    hipSetDevice(0);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time1 = 0;
    float time2 = 0;



    for (int iteration = 0; iteration < REP+WARMUP; iteration++){

      // printf("%f, %d,\n", input,iteration);            
      // Initialize the host input vectors
      thrust::host_vector<T> h_vec(size);
      thrust::detail::normal_iterator<T *> h_A = h_vec.begin();

      srand(2014);
      for(int i = 0; i < numElements; i++)
        h_A[i] = i % 2 != 0 ? i:i+1;
      int M = (numElements * input)/100;
      int m = M;
      while(m>0){
        int x = (int)(numElements*(((float)rand()/(float)RAND_MAX)));
        if(h_A[x] % 2 != 0){
            h_A[x] = x * 2;
            m--;
        }
      }

  
      thrust::device_vector<int> d_vec = h_vec;
      thrust::device_vector<int> s_vec = h_vec;

      //measure performance
        
      hipEventRecord( start, 0 );

      auto new_end = thrust::remove_if(d_vec.begin(), d_vec.end(), is_even());

      hipDeviceSynchronize();
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&time1, start, stop);
        if(iteration >= WARMUP) time2 += time1;
            
        if(iteration == REP+WARMUP-1){
          float timer = time2 / REP;
          double bw = (double)((2 * numElements) * sizeof(T)) / (double)(timer * 1000000.0);
          printf("%d, %f, %f\n", input, timer, bw);
	}   
       }
    }
    return 0;
}

