#include <stdio.h>
#include <time.h>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

#define T int
#define WARMUP 2
#define REP 10

int main()
{
	for (float i = 0.0; i < 1.05; i += 0.1)
	{
		float input = i;
		int numElements = 16000000;
		size_t size = numElements * sizeof(T);

		int value = 0;
		hipSetDevice(0);

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		float time1 = 0;
		float time2 = 0;

		for (int iteration = 0; iteration < REP + WARMUP; iteration++)
		{

			thrust::host_vector<T> h_vec(size);
			thrust::detail::normal_iterator<T *> h_A = h_vec.begin();

			srand(2014);
			for (int i = 0; i < numElements; i++)
				h_A[i] = value;
			int M = (numElements * input) / 100;
			int m = M;
			while (m > 0)
			{
				int x = (int)(numElements * (((float)rand() / (float)RAND_MAX)));
				if (h_A[x] == value)
				{
					h_A[x] = x + 2;
					m--;
				}
			}

			thrust::device_vector<int> d_vec = h_vec;
			thrust::device_vector<int> s_vec = h_vec;

			hipEventRecord(start, 0);

			thrust::copy_if(d_vec.begin(), d_vec.end(), s_vec.begin(), is_even());

			hipDeviceSynchronize();
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&time1, start, stop);
			if (iteration >= WARMUP)
				time2 += time1;

			if (iteration == REP + WARMUP - 1)
			{
				float timer = time2 / REP;
				double bw = (double)((2 * numElements) * sizeof(T)) / (double)(timer * 1000000.0);
				printf("%f, %f, %f\n", input, timer, bw);
			}
		}
	}
	return 0;
}
